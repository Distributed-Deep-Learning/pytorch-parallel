
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void d_sigmoid(float *data){
  *data = (1.0 - *data) * *data;
}

int main(){
  float *d_data, h_data = 0;
  hipMalloc((void **)&d_data, sizeof(float));
  hipMemcpy(d_data, &h_data, sizeof(float), hipMemcpyHostToDevice);
  d_sigmoid<<<1,1>>>(d_data);
  hipMemcpy(&h_data, d_data, sizeof(float), hipMemcpyDeviceToHost);
  printf("data = %d\n", h_data);
  return 0;
}